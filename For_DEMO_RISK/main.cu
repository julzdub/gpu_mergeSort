#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <float.h>
#include "cpuMergeSort.h"
#include "timing.h"
#include "Kernels.h"

void copyArray(int* original, int* copy, int size);
void printOutput(FILE * f_out, int * arr, int n);

int willPrint = 0;

void usage()
{
	printf("Usage: <./fileName> <arraySize> <printOptional>");
}

void printOutput(FILE * f_out, int * arr, int n) {
   int i;
   
   for(i = 0; i < n; i ++)
      fprintf(f_out, "%d ", arr[i]);

   
}

int * fillArray(int n, int upbound)
{
   int i;
   
   int *ret = (int *)malloc(sizeof(int) * n );

   /* Intializes random number generator */
   //seeds the random number generator used by the function rand.
   srand(time(NULL));

   /* generate n random numbers from 0 to unbound - 1 */
   for( i = 0 ; i < n ; i++ ) {
      int num = (rand() % 100);
      ret[i] = num;
      //printf("%f\n", num);
   }
   return ret;
}

void runCPU(int * inputArray, int start, int end) {
	
    	clock_t now, then;
    	
    	printf("Timing CPU implementation…\n");
    	then = clock();
    	mergeSort(inputArray, start, end);
    	now =  clock();
    	
    	// measure the time spent on CPU
       float time = 0;
       time = timeCost(then, now);
       
       

       printf("CPU time cost in second: %f\n", time);
}

void copyArray(int* original, int* copy, int size) {
   int i;
   for(i = 0; i < size; i++){

      copy[i] = original[i];
   }
}


int main(int argc, char *argv[])
{
        FILE * output_CPU = NULL;
        output_CPU = fopen("output_CPU.txt", "w");
        FILE * output_GPU = NULL;
        output_GPU = fopen("output_GPU.txt", "w");
	
	//get user input (number of elements and printing option p)
	if(argc < 2 || argc > 3)
	{
		usage();
		return 1;
	}
	else if(argc == 2)
	{
		willPrint = 0;
	}
	else if(argv[2][0] == 'p')
	{
		willPrint = 1;
	}
	else
	{
		usage();
		return 1;
	}
	
	int arraySize = atoi(argv[1]);
	
	if(!arraySize)
	{
		printf("Array Size Too Large");
		usage();
		return 1;
	}
	
	//Create and fill input array
	int *cpuArray = fillArray(arraySize, 200);
	int *gpuArray = (int*)malloc(sizeof(int) *arraySize);
	copyArray(cpuArray, gpuArray, arraySize);

	//Call cpu setup
	runCPU(cpuArray, 0, arraySize - 1);
	printOutput(output_CPU, cpuArray, arraySize);

	
	//Call kernel setup
	printf("Timing CPU implementation…\n");
	int * result = runCUDA(gpuArray, arraySize);
    printOutput(output_GPU, result, arraySize);
    	
    	if(willPrint) {
		printArray(cpuArray, arraySize);
		printArray(result, arraySize);
	}
	//Call cpu setup


	free(cpuArray);
	free(result);

	return 0;
}
