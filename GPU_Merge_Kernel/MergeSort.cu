#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

//TEMP HEADER
void printArray(int *arr, int n);
void runCUDA( int *arr, int n, int tile_width);
__global__ void mergeSortKernel(double *in, int n);
int * fillArray(int n, int upbound);
inline void check_cuda_errors(const char *filename, const int line_number);
int validateOutput(int size, int tile_width, int *arr);
void printArray2(int *arr,int start);
//TEMP HEADER


inline void check_cuda_errors(const char *filename, const int line_number)
{
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        printf("CUDA error at %s:%i: %s\n", filename, line_number, hipGetErrorString(error));
        exit(-1);
    }
}

int * fillArray(int n, int upbound)
{
   int i;
   
   int *ret = (int *)malloc(sizeof(int) * n );

   /* Intializes random number generator */
   //seeds the random number generator used by the function rand.
   srand(time(NULL));

   /* generate n random numbers from 0 to unbound - 1 */
   for( i = 0 ; i < n ; i++ ) {
      int num = (rand() % 100);
      ret[i] = num;
      //printf("%f\n", num);
   }
   return ret;
}

__global__ void mergeSortKernel(int *in, int *out, int n)
{
	extern __shared__ int sdata[];
	
	// load the shared memory
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

   if(i < n){
      sdata[tid] = in[i];
   }

   if(i+blockDim.x < n){
      sdata[tid+blockDim.x] = in[i+blockDim.x] ;
   }
   __syncthreads();


   for(unsigned int s = 2; s <= (blockDim.x*2); s*=2){
        if(tid < (blockDim.x*2)/s && s > 2){
            int start1 = tid*s; //First array offset
            int start2 = start1+(s/2); //Second array offset
            int size = s/2; //Size of each array will be the stride cut in half

            for(int x = size-1; x>=0; x--){
               int j;
               int last = sdata[start2-1];
         

               for(j = size-2; j>= 0 && sdata[start1+j] > sdata[start2+x]; j--){
                  sdata[start1+(j+1)] = sdata[start1+(j)];
               }

               if(j != size-2 || last > sdata[start2+x]){
                  sdata[start1+(j+1)] = sdata[start2+x];
                  sdata[start2+x] = last;
               }
            }
        }else if(tid < (blockDim.x*2)/s){ //The initial comparison of just two elements
           int start = tid*s;
           if(sdata[start] > sdata[start+1]){
              int temp = sdata[start];
              sdata[start] = sdata[start+1];
              sdata[start+1] = temp;
           }
        }
        __syncthreads();

}

   if(i < n){
      out[i] = sdata[tid];
   }

   if(i+blockDim.x < n){
      out[i+blockDim.x] = sdata[tid+blockDim.x] ;
   }


}

void runCUDA( int *arr, int n, int tile_width)
{    
   int *h_in = arr; //Filled array
   int *h_out = (int *)malloc(sizeof(int) * n ); //Allocate output mem
   int *d_in;//Device in pointer
   int *d_out;

   hipMalloc((void**)&d_in, sizeof(int) *n);//Allocate in and out mem
   hipMalloc((void**)&d_out, sizeof(int) *n);//Allocate in and out mem

   hipMemcpy(d_in, h_in, sizeof(int)*n, hipMemcpyHostToDevice);//Copy in array to device

   int num_block = (ceil(n/(double)tile_width));//Calculate grid size
   printf("\n\n\nArray size is %d\n", n);
   printf("Num of blocks is %d\n", num_block);
   printf("Tile size is %d\n", tile_width);
   printf("Active threads on run 1 are %f\n\n\n", ((double)tile_width)/2);
   dim3 block(tile_width/2,1,1);//Only 1/2 threads per block
   dim3 grid(num_block, 1,1);//Define grid

   //launch this shit and hope it works
   mergeSortKernel<<<grid, block, (tile_width)*sizeof(int)>>>(d_in, d_out, n);
   check_cuda_errors(__FILE__, __LINE__);

   hipMemcpy(h_out, d_out, sizeof(int)*n, hipMemcpyDeviceToHost);

   //printArray(h_out, n);
   
   int isVal = validateOutput(n, tile_width, h_out);

    if(isVal == -1){
       puts("\n\nOutput is valid!");
    }else{
       printf("\n\nOutput is NOT valid! At: %d\n\n", isVal);
       printArray2(h_out, isVal);
    }

   free(h_in);
   free(h_out);
   hipFree(d_in);
   hipFree(d_out);
}

int validateOutput(int size, int tile_width, int *arr){

   int i;
   for(i = 0; i < size/tile_width; i++){
      int j;
      for(j = 1; j < tile_width; j++){
         if(arr[(i*tile_width)+j] < arr[(i*tile_width)+j-1] && arr[(i*tile_width)+j] != arr[(i*tile_width)+j-1]){
            printf("\n\n%d > %d", arr[(i*tile_width)+j], arr[(i*tile_width)+j-1]);
            return (i*tile_width)+j;
         }
      }
   }
   return -1;

}


void printArray(int *arr, int n){

   int i;

   for(i = 0; i < n; i ++)
      printf("%d ", arr[i]);

   printf("\n");
}

//To print out elements arround an invalid output
void printArray2(int *arr,int start){

   int i;

   for(i = -25; i < 25; i ++)
      printf("%d ", arr[start+i]);

   printf("\n");
}


int main(int argc, char *argv[]){
   int size = 33554432; //Set Array size here
   int tile_width = 1024; //set tile_width here


   int* arr = fillArray(size, 10000);


    runCUDA( arr, size, tile_width); // Array, Elements, Tile size

}


