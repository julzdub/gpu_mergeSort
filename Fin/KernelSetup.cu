#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "MergingKernels.h"
#include "KernelSetup.h"

void runCuda(uint *array, uint *keys, uint dataSize, uint num_bytes)
{
	uint *inputKeys; 
	uint *outputKeys;
	uint *inputValues; 
	uint *outputValues;
	uint *keyResult;
	uint *valResult;
	
	//TODO: Make sure tile size is not larger than the array itself
	uint tileWidth = 4;
	
	
	//Allocating cuda memory for all arrays
	keyResult = (uint*)malloc(num_bytes);
	valResult = (uint*)malloc(num_bytes);
	hipMalloc((void**)&inputKeys, num_bytes);
	hipMalloc((void**)&outputKeys, num_bytes);
	hipMalloc((void**)&inputValues, num_bytes);
	hipMalloc((void**)&outputValues, num_bytes);
	
	if(0 == inputKeys || 0 == outputKeys || 0 == inputValues || 0 == outputValues)
	{
		printf("couldnt allocate memory\n");
		exit(-1);
	}
	
	//Setting number of blocks and threads
	uint numBlocks = ceil(dataSize / (uint)tileWidth);
	uint numThreads = SHARED_SIZE_LIMIT/2;
	printf("Number of blocks: %d\n", numBlocks);
	
	//Populating existing arrays
	hipMemcpy(inputKeys, array, num_bytes, hipMemcpyHostToDevice);
	hipMemcpy(inputValues, keys, num_bytes, hipMemcpyHostToDevice);
	hipMemset(outputKeys, 0, num_bytes);
	hipMemset(outputValues, 0, num_bytes);
	
	//Calling merging kernel
	mergeSortSharedKernel<1U><<<numBlocks, numThreads>>>(outputKeys, outputValues, inputKeys, inputValues, dataSize);
	
	//Capturing output from kernel
	hipMemcpy(keyResult, outputKeys, num_bytes, hipMemcpyDeviceToHost);
	hipMemcpy(valResult, outputValues, num_bytes, hipMemcpyDeviceToHost);
	
	
	printf("SUCCESSFUL\n");
	for(int i = 0; i < dataSize; i++)
	{
		printf("%d, %d\n", keyResult[i], valResult[i]);
	}
	
	hipFree(inputKeys);
	hipFree(inputValues);
	hipFree(outputKeys);
	hipFree(outputValues);
	free(keyResult);
	free(valResult);
}
