#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <float.h>
#include "timing.h"
#include "MergeSortSetup.h"
#include "Kernels.h"
  /////////////////////////////////////////////////////////////////////////////////////
 ////////////////////////////RUN CUDA SORT TILE FUNCTION//////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////

void runCUDA( int *array, int dataSize)
{  

   int * keys = createKeyArray(dataSize);
   int num_bytes = dataSize * sizeof(int);

   int *inputKeys; 
   int *outputKeys;
   int *inputValues; 
   int *outputValues;
   int *keyResult;
   int *valResult;

   int tileWidth = 1024;

   //Allocating cuda memory for all arrays
   keyResult = (int*)malloc(num_bytes);
	valResult = (int*)malloc(num_bytes);
	hipMalloc((void**)&inputKeys, num_bytes);
	hipMalloc((void**)&outputKeys, num_bytes);
	hipMalloc((void**)&inputValues, num_bytes);
	hipMalloc((void**)&outputValues, num_bytes);
	
	if(0 == inputKeys || 0 == outputKeys || 0 == inputValues || 0 == outputValues)
	{
		printf("couldnt allocate memory\n");
		exit(-1);
	}

   	//Setting number of blocks and threads
	//uint numBlocks = batchSize * dataSize / SHARED_SIZE_LIMIT;
   uint numBlocks =  dataSize / SHARED_SIZE_LIMIT;
	//uint numBlocks = ceil(dataSize / (double)tileWidth);
	uint numThreads =  SHARED_SIZE_LIMIT/2;

   //Populating existing arrays
	hipMemcpy(inputKeys, array, num_bytes, hipMemcpyHostToDevice);
	hipMemcpy(inputValues, keys, num_bytes, hipMemcpyHostToDevice);
	hipMemset(outputKeys, 0, num_bytes);
	hipMemset(outputValues, 0, num_bytes);

   hipEvent_t launch_begin, launch_end;
   hipEventCreate(&launch_begin);
   hipEventCreate(&launch_end);

   hipEventRecord(launch_begin, 0);
   //Calling merging kernel
	mergeSortSharedKernel<<<numBlocks, numThreads>>>(outputKeys, outputValues, inputKeys, inputValues, 1024, 1);
	hipEventRecord(launch_end, 0);
   hipEventSynchronize(launch_begin);
   hipEventSynchronize(launch_end);
   check_cuda_errors(__FILE__, __LINE__);
   hipDeviceSynchronize();
	//Capturing output from kernel
	hipMemcpy(keyResult, outputKeys, num_bytes, hipMemcpyDeviceToHost);
	hipMemcpy(valResult, outputValues, num_bytes, hipMemcpyDeviceToHost);

      //To record time at the end
   float time = 0;
   hipEventElapsedTime(&time, launch_begin, launch_end);
   printf("Done! Time cost is. %f\n", time/1000);

   //Destroy cudaEvents
   hipEventDestroy(launch_begin);
   hipEventDestroy(launch_end);

     	printf("SUCCESSFUL\n");
    int pk = 0;
    if(pk ==1){
        for(int i = 0; i < dataSize; i++)
	    {
		    printf("%d, %d\n", keyResult[i], valResult[i]);
	    }
    }

    int isVal = validateOutput(dataSize, 1024, keyResult);

    if(isVal == -1){
       puts("\n\nOutput is valid!");
    }else{
       printf("\n\nOutput is NOT valid! At: %d\n\n", isVal);
       printArray2(keyResult, isVal);
    }
//TODO:
	
	
	hipFree(inputKeys);
	hipFree(inputValues);
	hipFree(outputKeys);
	hipFree(outputValues);
	free(keyResult);
	free(valResult);

}


  /////////////////////////////////////////////////////////////////////////////////////
 ////////////////////////////RUN CUDA MERGE TILE FUNCTION/////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////


void runMerge(int * arr, int n, int tile_width, float time){
   int *h_in = arr; //Filled array
   int *h_out = (int *)malloc(sizeof(int) * n ); //Allocate output mem
   int *d_in;//Device in pointer
   int *d_out;

   hipMalloc((void**)&d_in, sizeof(int) *n);//Allocate in and out mem
   hipMalloc((void**)&d_out, sizeof(int) *n);//Allocate in and out mem

   hipMemcpy(d_in, h_in, sizeof(int)*n, hipMemcpyHostToDevice);//Copy in array to device
   
   int t_width = tile_width;
   int num_block;


   int* temp;

   hipEvent_t launch_begin, launch_end;
   hipEventCreate(&launch_begin);
   hipEventCreate(&launch_end);

   hipEventRecord(launch_begin, 0);
   while(t_width <= n/2){
      puts("\n\n RAN \n\n");
      num_block = n /(t_width * 2);
      mergeKernel<<<num_block,1>>>(d_in, d_out, n, t_width);
      t_width *= 2;
      
      temp = d_in;
      d_in = d_out;
      d_out = temp;
   }
   hipEventRecord(launch_end, 0);
   hipEventSynchronize(launch_begin);
   hipEventSynchronize(launch_end);

   //To record time at the end
   float time2 = 0;
   hipEventElapsedTime(&time2, launch_begin, launch_end);
   printf("Done! Time cost is. %f\n", time2/1000);

   //Destroy cudaEvents
   hipEventDestroy(launch_begin);

   hipMemcpy(h_out, d_in, sizeof(int)*n, hipMemcpyDeviceToHost);

   int isVal2 = validateOutput(n, n, h_out);

    if(isVal2 == -1){
       puts("\n\nOutput is valid!");
    }else{
       printf("\n\nOutput is NOT valid! At: %d\n\n", isVal2);
       printArray2(h_out, isVal2);
    }
    puts("\n\n\n\n\n");

    //printArray(h_out, n);

   hipFree(d_in);
   hipFree(d_out);
}


  /////////////////////////////////////////////////////////////////////////////////////
 /////////////////////////////////HELPER FUNCTIONS////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////


inline void check_cuda_errors(const char *filename, const int line_number)
{
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        printf("CUDA error at %s:%i: %s\n", filename, line_number, hipGetErrorString(error));
        exit(-1);
    }
}


int * createKeyArray(int size)
{
	int * array = (int*)malloc(sizeof(int) * size);
	for(int i = 0; i < size; i++)
	{
		array[i] = i;	
	}
	return array;
}


int validateOutput(int size, int tile_width, int *arr){

   int i;
   for(i = 0; i < size/tile_width; i++){
      int j;
      for(j = 1; j < tile_width; j++){
         if(arr[(i*tile_width)+j] < arr[(i*tile_width)+j-1] && arr[(i*tile_width)+j] != arr[(i*tile_width)+j-1]){
            printf("\n\n%d > %d", arr[(i*tile_width)+j], arr[(i*tile_width)+j-1]);
            return (i*tile_width)+j;
         }
      }
   }
   return -1;

}


void printArray(int *arr, int n){

   int i;

   for(i = 0; i < n; i ++)
      printf("%d ", arr[i]);

   printf("\n");
}


void printArray2(int *arr,int start){

   int i;

   for(i = -25; i < 25; i ++)
      printf("%d ", arr[start+i]);

   printf("\n");
}






















